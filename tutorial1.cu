#include "hip/hip_runtime.h"



#include "tutorial.h"
//#include <optix_device.h>
#include "random.h"
#include <stdlib.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>
//#include<random>


rtDeclareVariable(float3, shading_normal, attribute shading_normal, );	//all basic functions to create geometry occurs here,
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );//Defined functions can be used in everywhere  
rtDeclareVariable(int, primid, attribute primid, );

rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(unsigned int, radiance_ray_type, , );
rtDeclareVariable(float, scene_epsilon, , );
rtDeclareVariable(rtObject, top_object, , );
rtDeclareVariable(uint, launch_index1, rtLaunchIndex, );


static __device__ __inline__ float fold(const float value)
{
	return fminf(value, 1.0f - value) * 2.0f;
}



//
// Pinhole camera implementation
//
rtDeclareVariable(float3, eye, , );		//we use them in main code for calculation camera and visualization camera
rtDeclareVariable(float3, U, , );
rtDeclareVariable(float3, V, , );
rtDeclareVariable(float3, W, , );



rtDeclareVariable(float3, bad_color, , );
rtBuffer<uchar4, 2>              output_buffer;


rtBuffer<float3, 1>            normals;		//buffers provide temporary memory for us to be able to pass them to other variables
rtBuffer<float3, 1>            centers;
rtBuffer<float3, 1>            vertex_buffer;
//rtBuffer<float3, 1>            points;
//rtBuffer<float3, 1>            vertex_buffer2;
//rtBuffer<float, 1>             Energy;
rtBuffer<float, 1>             Energy_init;
rtBuffer<float, 1>             Extinction;
rtBuffer<float, 1>             hitting_prim;
rtBuffer<float, 1>             total_prim;
//rtBuffer<float, 1>             Energy_volume;
rtBuffer<float3, 1>            vertex_normal;
//rtBuffer<float3, 1>            vertex_normal2;
//rtBuffer<float3, 1>            hitting_normal;
//rtBuffer<float3, 1>             hitting_cord;
//rtBuffer<float, 2>              output_bufferx;
//rtBuffer<float, 2>              output_buffery;
//rtBuffer<float, 2>              output_bufferz;
rtBuffer<float, 2>              output_bufferl;
//rtBuffer<uint2, 1>              output_launch_index;
rtBuffer<int, 1>                  out;
rtBuffer<int, 1>                  out2;
rtBuffer<int, 1>                  out3;



rtBuffer<float2, 1>                  a;
rtBuffer<uint, 1>                  countbuffer;
rtDeclareVariable(float3, new_eye, , );
rtDeclareVariable(float3, new_normal, , );
rtDeclareVariable(int, ID, , );
rtDeclareVariable(int, count, , );
rtDeclareVariable(int, maxcount, , );
rtDeclareVariable(float, viewfactor, , );
rtDeclareVariable(int, totaltraingles, , );
rtDeclareVariable(int, flag, , );

//random yahan



RT_PROGRAM void random_camera()
{

	size_t2 screen = output_buffer.size();

	float2 d = make_float2(launch_index) / make_float2(screen) * 2.f - 1.f;
	float3 ray_origin = eye;							//ray source point
	float3 ray_direction = normalize(d.x*U + d.y*V + W);				// make unit position to real position

	optix::Ray ray(ray_origin, ray_direction, radiance_ray_type, scene_epsilon);	//its generation function

	PerRayData_radiance prd;	
	prd.importance = 1.f;
	prd.depth = 0;

	rtTrace(top_object, ray, prd);							//important function to trace and find intersection point

	output_buffer[launch_index] = make_color(prd.result);				//demonstrate result as different colors

}



RT_PROGRAM void pinhole_camera()
{

	PerRayData_radiance prd;


	// double coef = (pow(a, e));

	size_t2 screen = output_buffer.size();

	uint2 seed;
	seed.x = tea<16>(count + screen.x*launch_index.y + launch_index.x, (count + 100));
	seed.y = tea<16>(count + screen.y*launch_index.x + launch_index.y, (count + 100));
	uint seedz = tea<16>(2 + +screen.y*launch_index.x + launch_index.y, 112);

	uint seez = seed.x + seed.y;
	double r1 = rnd(seed.x);
	double r2 = rnd(seed.y);
	double r3 = rnd((seez));



	//prd.random = r1;

	int ran = 0;

	//hiprandState state;

	//hiprand_init(seed.x, 0, 0, &state);   // generate random numbers




	if (screen.x*launch_index.y + launch_index.x < totaltraingles)
	{
		ran = screen.x*launch_index.y + launch_index.x;//hiprand(&state) % totaltraingles;    //screen.x*launch_index.y + launch_index.x; //hiprand(&state) % totaltraingles; 



	}
	else
	{
		ran = (screen.x*launch_index.y + launch_index.x) % (totaltraingles - 1);//hiprand(&state) % ((totaltraingles));//(screen.x*launch_index.y + launch_index.x) % (totaltraingles - 1); //hiprand(&state) % ((totaltraingles - 1));// //(screen.x*launch_index.y + launch_index.x) % (totaltraingles);//  //// / 

	}
	//rtPrintf("rand is %d", ran);


	float ph = 0 * M_PI / 180;    //180
	float th = (90)*M_PI / 180; //60



	float3 W_normal = normals[ran]; //make_float3(0.000001, 1, 0.0000001);//normals[ran]; //make_float3(sin(th)*sin(ph), cos(th), -sin(th)*cos(ph));//normals[screen.x*launch_index.y + launch_index.x]; // normals[count];// normals[ran];// vertex_normal[count]; //normals[count];////normals[ran];//new_normal;//normals[count];//new_normal;//vertex_normal[count]; // new_normal //new_normal



	float3 up_new = make_float3(0, 1, 0);
	up_new = normalize(up_new);

	float3 U_new = cross(W_normal, up_new);
	float3 V_new = cross(U_new, W_normal);




	//random




	double rad = length(W_normal);   //W_normal

	float3 rx = (((rad*normalize((U_new))))); //U_new  
	float3 ry = (rad*normalize(V_new));   // V_new
	float3 rz = (rad*normalize(W_normal)); //W_new

	// mapping into hemisphere

	float ee = 0;
	float cos_phi = cos((2 * M_PI)*r1);                 // for whole sphere cos( (M_PI)*d.x); where d.x is random from -1 to 1
	float sin_phi = sin((2 * M_PI)*r1);                 // // for hemisohere sphere cos( (2*M_PI)*r1); where 1 is random from 0 to 1
	//float cos_theta = (((1 - 2 * r2)));                       // remove 2 fro original method
	double sin_theta = sqrt(r2);
	float cos_theta = sqrt(1 - (sin_theta*sin_theta));  // from pinar hoca's book
	// for new try
	//float sin_theta = sqrt(1 - (cos_theta*cos_theta));  // cosine distribtion
	float px = sin_theta*cos_phi;//sin_theta*sin_phi;
	float py = sin_theta*sin_phi; //cos_theta;
	float pz = cos_theta;//sin_theta*cos_phi;


	float3 d_mapping = normalize(px*(rx)+py*(ry)+pz*(rz));



	//d_f

	//selecting random position within cube



	float3 ray_origin = centers[ran];// point;//make_float3(0.01f, 0.02f, 0.2f);//point;//centers[screen.x*launch_index.y + launch_index.x];// centers[count];// centers[count];// point;// //point;// centers[ran]; //point;// // centers[ran];  //vertex_buffer[count];// centers[ran];// new_eye;//vertex_buffer[count];  centers[count];// //new_eye for sampling



	int a = atomicAdd(&out[0], 1);  // counting generated rays

	atomicAdd(&total_prim[ran], 1);


	//hitting_prim[ran] = 0;

	prd.prim = ran;



	optix::Ray ray(ray_origin, d_mapping, radiance_ray_type, scene_epsilon); //ray_direction //d_mapping


	rtTrace(top_object, ray, prd);





	//	output_bufferl[launch_index] = ray_dot;

	countbuffer[0] = count;

	countbuffer[1] = maxcount;




}


//
// Returns solid color for miss rays
//


rtDeclareVariable(float3, bg_color, , );
RT_PROGRAM void miss()
{
	prd_radiance.result = bg_color;
	// atomicAdd(&Energy_init[output_buffer.size().x*launch_index.y + launch_index.x], (1.81 * 1000 / 24)*abs(dot(prd_radiance.normal, prd_radiance.direction)));
	//atomicAdd(&Energy_init[output_buffer.size().x*launch_index.y + launch_index.x], 1);

}

//
// (UPDATED)
// Implements basic lambertian surface shading model 
//
rtDeclareVariable(float3, Ka, , );
rtDeclareVariable(float3, Kd, , );

rtDeclareVariable(float3, ambient_light_color, , );
rtBuffer<BasicLight> lights;


RT_PROGRAM void closest_hit_radiance1()
{


	float3 hit_point = ray.origin + t_hit * ray.direction;
	float vf = 0.5;// float(hitting_prim[primid]) / float(total_prim[primid]);

	//rtPrintf("\n\n vf of prim id %f is %f", hitting_prim[6000], total_prim[6000]);

	float r = min(max(0.0f, 1.5 - abs(1 - 4 * (vf - 0.4))), 1.0f);
	float g = min(max(0.0f, 1.5 - abs(1 - 4 * (vf - 0.15))), 1.0f);
	float b = min(max(0.0f, 1.5 - abs(1 - 4 * vf)), 1.0f);


	prd_radiance.result = make_float3(r, g, b);  //Extinction[prim_index]
	//prd_radiance.result = make_float3(1.0, 0.0, 0.0);

}

RT_PROGRAM void closest_hit_radiance_object()
{

	//hitting_normal[count] = ffnormal;
	int countt = atomicAdd(&out3[0], 1);

	int id = prd_radiance.prim;


	atomicAdd(&hitting_prim[id], 1);

	prd_radiance.result = make_float3(1, 0, 0);

}


//
// Set pixel to solid color upon failure
//
RT_PROGRAM void exception()
{
	output_buffer[launch_index] = make_color(bad_color);
}

RT_PROGRAM void any_hit() // object
{




}


RT_PROGRAM void any_hit2()  // box_matl (source)
{


}

RT_PROGRAM void closest_hit_radiance_test()
{
	float3 world_geo_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometric_normal));
	float3 world_shade_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
	float3 ffnormal = faceforward(world_shade_normal, -ray.direction, world_geo_normal);
	float3 color = Ka * ambient_light_color;

	float3 hit_point = ray.origin + t_hit * ray.direction;

	for (int i = 0; i < lights.size(); ++i) {
		BasicLight light = lights[i];
		float3 L = normalize(light.pos - hit_point);
		float nDl = dot(ffnormal, L);

		if (nDl > 0.0f){
			// cast shadow ray
			PerRayData_shadow shadow_prd;
			shadow_prd.attenuation = make_float3(1.0f);
			float Ldist = length(light.pos - hit_point);
			//			optix::Ray shadow_ray(hit_point, L, shadow_ray_type, scene_epsilon, Ldist);
			//			rtTrace(top_shadower, shadow_ray, shadow_prd);
			float3 light_attenuation = shadow_prd.attenuation;

			if (fmaxf(light_attenuation) > 0.0f) {
				float3 Lc = light.color * light_attenuation;
				color += Kd * nDl * Lc;

				float3 H = normalize(L - ray.direction);
				float nDh = dot(ffnormal, H);
				//if (nDh > 0)
				//	color += 0.3 * Lc * pow(nDh, 0.3);  //phong // ks 0.3
			}
		}
	}
	prd_radiance.result = color;
}



RT_PROGRAM void closest_hit_radiance_test2()
{
	float3 world_geo_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometric_normal));
	float3 world_shade_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
	float3 ffnormal = faceforward(world_shade_normal, -ray.direction, world_geo_normal);
	float3 color = Ka * make_float3(0.5, 0.5, 4.0);
	float3 hit_point = ray.origin + t_hit * ray.direction;

	for (int i = 0; i < lights.size(); ++i) {
		BasicLight light = lights[i];
		float3 L = normalize(light.pos - hit_point);
		float nDl = dot(ffnormal, L);

		if (nDl > 0.0f){
			PerRayData_shadow shadow_prd;
			shadow_prd.attenuation = make_float3(1.0f);
			float Ldist = length(light.pos - hit_point);
			float3 light_attenuation = shadow_prd.attenuation;
			if (fmaxf(light_attenuation) > 0.0f)
			{
				float3 Lc = light.color * light_attenuation;
				color += Kd * nDl * Lc;
				float3 H = normalize(L - ray.direction);
				float nDh = dot(ffnormal, H);

			}
		}
	}

	prd_radiance.result = color;

}


// camera spherical
// spherical camera
/*'
double rt = sqrt(d_centers.x* d_centers.x + d_centers.y* d_centers.y);
//if ((rt*rt) <= 1)
{


double SI = rt*(M_PIf / 2);  //si max should be pi/2//
double sa = d_centers.y / rt;
double ca = d_centers.x / rt;
float3 d_spherical = normalize(sin(SI)*ca* (rx)+sin(SI)*sa* (ry)+cos(SI)*(rz));

float rt_nodes = sqrt(d_nodes.x* d_nodes.x + d_nodes.y* d_nodes.y);

// nodes
float SI_nodes = rt_nodes*(M_PI / 2);
float sa_cord = d_nodes.y / rt_nodes;
float ca_cord = d_nodes.x / rt_nodes;

float3 d_spherical_cord = eye + (sin(SI_nodes)*ca_cord* (rx)+sin(SI_nodes)*sa_cord* (ry)+cos(SI_nodes)*(rz));


//camera spherical

double lambda = d_centers.x* (M_PIf / 2);
double si = d_centers.y* (M_PIf / 2);


double phi = M_PIf - lambda;
double theta = (M_PIf / 2) - si;

float3 d_2 = normalize(sin(theta)*sin(phi)*(rx)+cos(theta)*(ry)-sin(theta)*cos(phi)*(rz)); */


